#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>
#include "../../matrices/matrix_utils.h"

using namespace std;
using namespace std::chrono;

// CUDA Kernel pour la multiplication des matrices `float`
__global__
void multiplyMatricesFloat(float *mat1, float *mat2, float *result, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        float sum = 0.0;
        for (int k = 0; k < N; k++) {
            sum += mat1[row * N + k] * mat2[k * N + col];
        }
        result[row * N + col] = sum;
    }
}

// CUDA Kernel pour la multiplication des matrices `int`
__global__
void multiplyMatricesInt(int *mat1, int *mat2, int *result, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < N; k++) {
            sum += mat1[row * N + k] * mat2[k * N + col];
        }
        result[row * N + col] = sum;
    }
}


int main(int argc, char *argv[]) {
    if (argc != 3) {
        cerr << "Utilisation : " << argv[0] << " <fichier_matrice1.csv> <fichier_matrice2.csv>" << endl;
        return EXIT_FAILURE;
    }

    const char *fichier1 = argv[1];
    const char *fichier2 = argv[2];

    bool is_float = type_matrice(fichier1) || type_matrice(fichier2);
    int taille1, taille2;

    void *h_mat1 = nullptr, *h_mat2 = nullptr, *h_result = nullptr;
    void *d_mat1 = nullptr, *d_mat2 = nullptr, *d_result = nullptr;

    // Charger les matrices
    charger_matrice_csv(fichier1, &h_mat1, &taille1, is_float);
    charger_matrice_csv(fichier2, &h_mat2, &taille2, is_float);

    if (taille1 != taille2) {
        cerr << "Erreur : Les matrices doivent avoir la même taille." << endl;
        free(h_mat1);
        free(h_mat2);
        return EXIT_FAILURE;
    }

    int N = taille1;
    int matrixSize = N * N * (is_float ? sizeof(float) : sizeof(int));

    int blockSize = 256;
    int numBlocks = (N * N + blockSize - 1) / blockSize;

    h_result = malloc(matrixSize);
    if (!h_result) {
        cerr << "Erreur d'allocation mémoire sur l'hôte." << endl;
        free(h_mat1);
        free(h_mat2);
        return EXIT_FAILURE;
    }

    auto start = high_resolution_clock::now();

    hipMalloc(&d_mat1, matrixSize);
    hipMalloc(&d_mat2, matrixSize);
    hipMalloc(&d_result, matrixSize);

    hipMemcpy(d_mat1, h_mat1, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_mat2, h_mat2, matrixSize, hipMemcpyHostToDevice);

    if (is_float) {
        multiplyMatricesFloat<<<numBlocks, blockSize>>>((float*)d_mat1, (float*)d_mat2, (float*)d_result, N);
    } else {
        multiplyMatricesInt<<<numBlocks, blockSize>>>((int*)d_mat1, (int*)d_mat2, (int*)d_result, N);
    }
    hipDeviceSynchronize();

    hipMemcpy(h_result, d_result, matrixSize, hipMemcpyDeviceToHost);

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(stop - start);

    cout << "Multiplication terminée en " << duration.count() << " ms sur GPU (CUDA)." << endl;

    // char nom_fichier[256];
    // generer_nom_fichier_resultat(nom_fichier, sizeof(nom_fichier), "res/cuda", "mul", is_float, N);
    // sauvegarder_matrice_csv(nom_fichier, h_result, N, is_float);
    // cout << "Résultat enregistré dans : " << nom_fichier << endl;

    free(h_mat1);
    free(h_mat2);
    free(h_result);
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_result);


    return EXIT_SUCCESS;
}
