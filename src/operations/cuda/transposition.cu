#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>
#include "../../matrices/matrix_utils.h"

using namespace std;
using namespace std::chrono;

// CUDA Kernel pour la transposition d'une matrice `float`
__global__
void transpositionMatrixFloat(float *mat, float *result, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        result[col * N + row] = mat[row * N + col];
    }
}

// CUDA Kernel pour la transposition d'une matrice `int`
__global__
void transpositionMatrixInt(int *mat, int *result, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        result[col * N + row] = mat[row * N + col];
    }
}


int main(int argc, char *argv[]) {
    if (argc < 2) {
        cerr << "Utilisation : " << argv[0] << " <fichier_matrice1.csv>" << endl;
        return EXIT_FAILURE;
    }

    const char *fichier1 = argv[1];
    bool is_float = type_matrice(fichier1);
    int taille;
    void *h_mat = nullptr, *h_result = nullptr;
    void *d_mat = nullptr, *d_result = nullptr;

    // Charger la matrice
    charger_matrice_csv(fichier1, &h_mat, &taille, is_float);

    h_result = malloc(is_float ? sizeof(float) : sizeof(int));
    if (!h_result) {
        cerr << "Erreur d'allocation mémoire" << endl;
        free(h_mat);
        return EXIT_FAILURE;
    }

    auto start = high_resolution_clock::now();
    hipMalloc(&d_mat, taille * taille * (is_float ? sizeof(float) : sizeof(int)));
    hipMalloc(&d_result, is_float ? sizeof(float) : sizeof(int));
    hipMemcpy(d_mat, h_mat, taille * taille * (is_float ? sizeof(float) : sizeof(int)), hipMemcpyHostToDevice);

    int blockSize = 256;
    if (is_float) {
        transpositionMatrixFloat<<<1, blockSize>>>((float*)d_mat, (float*)d_result, taille);
    } else {
        transpositionMatrixInt<<<1, blockSize>>>((int*)d_mat, (int*)d_result, taille);
    }
    hipDeviceSynchronize();
    hipMemcpy(h_result, d_result, is_float ? sizeof(float) : sizeof(int), hipMemcpyDeviceToHost);
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(stop - start);

    // cout << "Trace terminée en " << duration.count() << " ms sur GPU (CUDA)." << endl;
    // cout << "Résultat de la trace : " << (is_float ? *(float*)h_result : *(int*)h_result) << endl;

    free(h_mat);
    free(h_result);
    hipFree(d_mat);
    hipFree(d_result);

    return EXIT_SUCCESS;
}
