#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>
#include "../../matrices/matrix_utils.h"

using namespace std;
using namespace std::chrono;

// CUDA Kernel pour calculer le déterminant d'une matrice `float`
__global__
void determinantMatrixFloat(float *mat, float *result, int N) {
    int k = threadIdx.x; // Chaque thread représente une ligne

    __shared__ float tempMat[1024]; // Utilisation de mémoire partagée pour stocker la matrice temporaire

    // Charger la matrice en mémoire partagée
    if (k < N) {
        for (int j = 0; j < N; j++) {
            tempMat[k * N + j] = mat[k * N + j];
        }
    }
    __syncthreads();

    // Élimination de Gauss
    for (int i = 0; i < N; i++) {
        if (k > i) {
            float facteur = tempMat[k * N + i] / tempMat[i * N + i];
            for (int j = i; j < N; j++) {
                tempMat[k * N + j] -= facteur * tempMat[i * N + j];
            }
        }
        __syncthreads();
    }

    // Calcul du produit des éléments diagonaux
    if (k == 0) {
        float det = 1.0;
        for (int i = 0; i < N; i++) {
            det *= tempMat[i * N + i];
        }
        *result = det; // Stocker le résultat dans la mémoire globale
    }
}

// CUDA Kernel pour calculer le déterminant d'une matrice `int`
__global__
void determinantMatrixInt(int *mat, int *result, int N) {
    int k = threadIdx.x; // Chaque thread représente une ligne

    __shared__ int tempMat[1024]; // Utilisation de mémoire partagée pour stocker la matrice temporaire

    // Charger la matrice en mémoire partagée
    if (k < N) {
        for (int j = 0; j < N; j++) {
            tempMat[k * N + j] = mat[k * N + j];
        }
    }
    __syncthreads();

    // Élimination de Gauss
    for (int i = 0; i < N; i++) {
        if (k > i) {
            int facteur = tempMat[k * N + i] / tempMat[i * N + i];
            for (int j = i; j < N; j++) {
                tempMat[k * N + j] -= facteur * tempMat[i * N + j];
            }
        }
        __syncthreads();
    }

    // Calcul du produit des éléments diagonaux
    if (k == 0) {
        int det = 1;
        for (int i = 0; i < N; i++) {
            det *= tempMat[i * N + i];
        }
        *result = det; // Stocker le résultat dans la mémoire globale
    }
}



int main(int argc, char *argv[]) {
    if (argc < 2) {
        cerr << "Utilisation : " << argv[0] << " <fichier_matrice1.csv>" << endl;
        return EXIT_FAILURE;
    }

    const char *fichier1 = argv[1];
    bool is_float = type_matrice(fichier1);
    int taille;
    void *h_mat = nullptr, *h_result = nullptr;
    void *d_mat = nullptr, *d_result = nullptr;

    // Charger la matrice
    charger_matrice_csv(fichier1, &h_mat, &taille, is_float);

    h_result = malloc(is_float ? sizeof(float) : sizeof(int));
    if (!h_result) {
        cerr << "Erreur d'allocation mémoire" << endl;
        free(h_mat);
        return EXIT_FAILURE;
    }

    auto start = high_resolution_clock::now();
    hipMalloc(&d_mat, taille * taille * (is_float ? sizeof(float) : sizeof(int)));
    hipMalloc(&d_result, is_float ? sizeof(float) : sizeof(int));
    hipMemcpy(d_mat, h_mat, taille * taille * (is_float ? sizeof(float) : sizeof(int)), hipMemcpyHostToDevice);

    int blockSize = 256;
    if (is_float) {
        determinantMatrixFloat<<<1, blockSize>>>((float*)d_mat, (float*)d_result, taille);
    } else {
        determinantMatrixInt<<<1, blockSize>>>((int*)d_mat, (int*)d_result, taille);
    }
    hipDeviceSynchronize();
    hipMemcpy(h_result, d_result, is_float ? sizeof(float) : sizeof(int), hipMemcpyDeviceToHost);
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(stop - start);

    // cout << "Determinant terminée en " << duration.count() << " ms sur GPU (CUDA)." << endl;
    // cout << "Résultat de la trace : " << (is_float ? *(float*)h_result : *(int*)h_result) << endl;

    free(h_mat);
    free(h_result);
    hipFree(d_mat);
    hipFree(d_result);

    return EXIT_SUCCESS;
}
